#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvToolsExtCuda.h>

#include <cstdlib>
#include <ctime>
#include <iostream>

#include "common/result_helper.h"

constexpr int K_THREAD_PER_BLOCK = 256;

__global__ void reduce3(float* d_in, float* d_out) {
  __shared__ float s_data[K_THREAD_PER_BLOCK];

  // each thread loads one element from global memory to shared memory
  const auto tid = threadIdx.x;
  const auto i = blockIdx.x * (2 * blockDim.x) + tid;
  s_data[tid] = d_in[i] + d_in[i + blockDim.x];
  __syncthreads();

  // do reduction in shared memory
  // int size = blockDim.x;
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    // if (tid + stride < size) {
    if (tid < stride) {
      s_data[tid] += s_data[tid + stride];
    }
    // size >>= 1;
    __syncthreads();
  }

  // write result for this block to global memory
  if (tid == 0) {
    d_out[blockIdx.x] = s_data[0];
  }
}

int main() {
  srand(time(0));
  const int N = 32 * 1024 * 1024;
  auto* a = new float[N];
  float* d_a;
  hipMalloc(&d_a, N * sizeof(float));

  const int thread_num = 2 * K_THREAD_PER_BLOCK;
  const int block_num = N / thread_num;
  auto* out = new float[block_num];
  float* d_out;
  hipMalloc(&d_out, block_num * sizeof(float));
  auto* res = new float[block_num];

  for (int i = 0; i < N; i++) {
    a[i] = rand() % 10000;
  }

  for (int i = 0; i < block_num; ++i) {
    float cur = 0;
    for (int j = 0; j < thread_num; ++j) {
      cur += a[i * thread_num + j];
    }
    res[i] = cur;
  }

  hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
  dim3 block(K_THREAD_PER_BLOCK, 1);
  dim3 grid(block_num, 1);

  nvtxRangePushA("reduce0");
  reduce3<<<grid, block>>>(d_a, d_out);
  nvtxRangePop();

  hipMemcpy(out, d_out, block_num * sizeof(float), hipMemcpyDeviceToHost);
  if (CheckResultSame(out, res, block_num)) {
    std::cout << "The answer is correct!" << std::endl;
  } else {
    std::cout << "The answer is wrong!" << std::endl;
    for (int i = 0; i < block_num; ++i) {
      std::cout << "out[" << i << "] = " << out[i] << " res[" << i
                << "] = " << res[i] << std::endl;
    }
    std::cout << "\n";
  }

  hipFree(d_a);
  hipFree(d_out);
  hipDeviceReset();

  delete[] out;
  delete[] res;
  delete[] a;
  return 0;
}